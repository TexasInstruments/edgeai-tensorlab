#include "hip/hip_runtime.h"
#include <torch/library.h>
// Copied and adapted from
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <ATen/native/cuda/UpSample.cuh>

// Below is experimental temporary code before merging it to PyTorch
namespace at {
namespace native {
namespace internal_upsample {

__device__ __forceinline__ size_t
idx(const size_t nc,
    const size_t height,
    const size_t width,
    const size_t y,
    const size_t x) {
  return (nc * height + y) * width + x;
}

// taken from
// https://github.com/python-pillow/Pillow/blob/6812205f18ca4ef54372e87e1a13ce4a859434df/
// src/libImaging/Resample.c#L20-L29
template <typename accscalar_t>
__device__ __forceinline__ static accscalar_t bilinear_filter(accscalar_t x) {
  if (x < 0.0) {
    x = -x;
  }
  if (x < 1.0) {
    return static_cast<accscalar_t>(1.0) - x;
  }
  return static_cast<accscalar_t>(0.0);
}

// taken from
// https://github.com/python-pillow/Pillow/blob/6812205f18ca4ef54372e87e1a13ce4a859434df/
// src/libImaging/Resample.c#L46-L62
template <typename accscalar_t>
__device__ __forceinline__ static accscalar_t bicubic_filter(accscalar_t x) {
  // https://en.wikipedia.org/wiki/Bicubic_interpolation#Bicubic_convolution_algorithm
#define a -0.5
  if (x < 0.0) {
    x = -x;
  }
  if (x < 1.0) {
    return ((a + 2.0) * x - (a + 3.0)) * x * x + static_cast<accscalar_t>(1.0);
  }
  if (x < 2.0) {
    return (((x - 5) * x + 8) * x - 4) * a;
  }
  return static_cast<accscalar_t>(0.0);
#undef a
}

template <typename scalar_t, typename accscalar_t, typename filter_fn_t>
__device__ __forceinline__ static void _compute_weights(
    const int i,
    const int input_size,
    const accscalar_t scale,
    const accscalar_t support,
    scalar_t* wt_ptr,
    int interp_size,
    filter_fn_t filter_fn,
    int& xmin,
    int& xmax) {
  accscalar_t invscale = (scale >= 1.0) ? 1.0 / scale : 1.0;
  accscalar_t center = scale * (i + 0.5);
  xmin = max(static_cast<int>(center - support + 0.5), static_cast<int>(0));
  xmax = min(static_cast<int>(center + support + 0.5), input_size) - xmin;

  accscalar_t total_w = 0.0;
  int j = 0;
  for (j = 0; j < xmax; j++) {
    accscalar_t w = filter_fn((j + xmin - center + 0.5) * invscale);
    wt_ptr[j] = static_cast<scalar_t>(w);
    total_w += w;
  }
  for (j = 0; j < xmax; j++) {
    if (total_w != 0.0) {
      wt_ptr[j] /= total_w;
    }
  }
  for (; j < interp_size; j++) {
    wt_ptr[j] = static_cast<scalar_t>(0.0);
  }
}

template <typename scalar_t, typename accscalar_t>
__device__ __forceinline__ static accscalar_t interpolate_aa_single_dim(
    scalar_t* src,
    scalar_t* weights,
    int64_t size) {
  scalar_t t = static_cast<accscalar_t>(*src);
  scalar_t wts = static_cast<accscalar_t>(weights[0]);
  accscalar_t output = t * wts;

  int64_t j = 1;
  for (; j < size; j++) {
    wts = static_cast<accscalar_t>(weights[j]);
    t = static_cast<accscalar_t>(*(src + j));
    output += t * wts;
  }
  return output;
}

template <typename scalar_t, typename accscalar_t, int interp_size>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_gen2d_out_frame(
    const int n,
    const accscalar_t rheight,
    const accscalar_t rwidth,
    const bool align_corners,
    const PackedTensorAccessor64<scalar_t, 4> idata,
    PackedTensorAccessor64<scalar_t, 4> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int height1 = idata.size(2);
  const int width1 = idata.size(3);
  const int height2 = odata.size(2);
  const int width2 = odata.size(3);

  if (index < n) {
    const int w2 = index % width2; // 0:width2-1
    const int h2 = index / width2; // 0:height2-1
    // special case: just copy
    if (height1 == height2 && width1 == width2) {
      const int h1 = h2;
      const int w1 = w2;
      for (int n = 0; n < batchsize; n++) {
        for (int c = 0; c < channels; ++c) {
          const scalar_t val = idata[n][c][h1][w1];
          odata[n][c][h2][w2] = val;
        }
      }
      return;
    }

    const accscalar_t support_h = static_cast<accscalar_t>(
        (rheight >= 1.0) ? (interp_size * 0.5) * rheight : interp_size * 0.5);
    const accscalar_t support_w = static_cast<accscalar_t>(
        (rwidth >= 1.0) ? (interp_size * 0.5) * rwidth : interp_size * 0.5);

    const int interp_height = (int)ceilf(support_h) * 2 + 1;
    const int interp_width = (int)ceilf(support_w) * 2 + 1;

    // Setup local buffers
    // TODO: maybe we can specify dynamic shared memory size before calling the
    // cuda code, however we should then ensure that device has enough shared
    // memory
    scalar_t wx[256];
    scalar_t wy[256];
    scalar_t buffer1[256];
    scalar_t buffer2[256];

    // Compute weights
    int xmin, xsize, ymin, ysize;
    typedef scalar_t (*filter_fn_t)(scalar_t);
    filter_fn_t filter_fn;
    if (interp_size == 2) {
      filter_fn = bilinear_filter;
    } else if (interp_size == 4) {
      filter_fn = bicubic_filter;
    }
    _compute_weights<scalar_t, accscalar_t, filter_fn_t>(
        w2,
        width1,
        rwidth,
        support_w,
        wx,
        interp_width,
        filter_fn,
        xmin,
        xsize);
    _compute_weights<scalar_t, accscalar_t, filter_fn_t>(
        h2,
        height1,
        rheight,
        support_h,
        wy,
        interp_height,
        filter_fn,
        ymin,
        ysize);

    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        // interpolate on x-axis for ymin to ymin + ysize
        for (int y = 0; y < ysize; y++) {
          // copy data into the local buffer and use
          // interpolate_aa_single_dim method
          for (int x = 0; x < xsize; x++) {
            buffer1[x] = idata[n][c][ymin + y][xmin + x];
          }

          buffer2[y] = static_cast<scalar_t>(
              interpolate_aa_single_dim<scalar_t, accscalar_t>(
                  buffer1, wx, xsize));
        }
        odata[n][c][h2][w2] = static_cast<scalar_t>(
            interpolate_aa_single_dim<scalar_t, accscalar_t>(
                buffer2, wy, ysize));
      }
    }
  }
}

template <int interp_size>
static void upsample_gen2d_out_cuda_template(
    const Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  // Copied and adapted from
  // UpSampleBicubic2d.cu::upsample_bicubic2d_out_cuda_template
  TensorArg input_arg{input, "input", 1}, output_arg{output, "output", 2};
  checkAllSameGPU("upsample_gen2d_out_cuda", {input_arg, output_arg});

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input.size(0);
  int channels = input.size(1);
  int input_height = input.size(2);
  int input_width = input.size(3);

  const int num_kernels = output_height * output_width;
  const int num_threads = std::min(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "upsample_gen2d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.packed_accessor64<scalar_t, 4>();
        auto odata = output.packed_accessor64<scalar_t, 4>();

        const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
            input_height, output_height, align_corners, scales_h);
        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners, scales_w);

        // We are using static buffer memory of 256 * sizeof(float) per thread
        // to store weights. Size of weights array is
        // interp_size = scale * 2 + 1 for bilinear mode
        TORCH_CHECK(
            rheight < (255 / interp_size),
            "Max supported scale factor is 127 (bilinear), 63 (bicubic)");
        TORCH_CHECK(
            rwidth < (255 / interp_size),
            "Max supported scale factor is 127 (bilinear), 63 (bicubic)");

        upsample_gen2d_out_frame<scalar_t, accscalar_t, interp_size>
            <<<cuda::ATenCeilDiv(num_kernels, num_threads),
               num_threads,
               0,
               stream>>>(
                num_kernels, rheight, rwidth, align_corners, idata, odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename scalar_t, typename accscalar_t, int interp_size>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_gen2d_backward_out_frame(
    const int num_elements,
    const accscalar_t height_scale,
    const accscalar_t width_scale,
    const bool align_corners,
    PackedTensorAccessor64<scalar_t, 4> idata,
    const PackedTensorAccessor64<scalar_t, 4> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int input_height = idata.size(2);
  const int input_width = idata.size(3);
  const int output_height = odata.size(2);
  const int output_width = odata.size(3);

  if (index >= num_elements) {
    return;
  }

  const int output_x = index % output_width;
  const int output_y = index / output_width;
  // special case: output just copy
  if (input_height == output_height && input_width == output_width) {
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const scalar_t val = odata[n][c][output_y][output_x];
        idata[n][c][output_y][output_x] = val;
      }
    }
    return;
  }

  const accscalar_t support_h = static_cast<accscalar_t>(
      (height_scale >= 1.0) ? (interp_size * 0.5) * height_scale
                            : interp_size * 0.5);
  const accscalar_t support_w = static_cast<accscalar_t>(
      (width_scale >= 1.0) ? (interp_size * 0.5) * width_scale
                           : interp_size * 0.5);

  const int interp_height = (int)ceilf(support_h) * 2 + 1;
  const int interp_width = (int)ceilf(support_w) * 2 + 1;

  // Setup local buffers
  // TODO: maybe we can specify dynamic shared memory size before calling the
  // cuda code, however we should then ensure that device has enough shared
  // memory
  scalar_t wx[256];
  scalar_t wy[256];

  // Compute weights
  int xmin, xsize, ymin, ysize;
  typedef scalar_t (*filter_fn_t)(scalar_t);
  filter_fn_t filter_fn;
  if (interp_size == 2) {
    filter_fn = bilinear_filter;
  } else if (interp_size == 4) {
    filter_fn = bicubic_filter;
  }
  _compute_weights<scalar_t, accscalar_t, filter_fn_t>(
      output_x,
      input_width,
      width_scale,
      support_w,
      wx,
      interp_width,
      filter_fn,
      xmin,
      xsize);
  _compute_weights<scalar_t, accscalar_t, filter_fn_t>(
      output_y,
      input_height,
      height_scale,
      support_h,
      wy,
      interp_height,
      filter_fn,
      ymin,
      ysize);

  for (int n = 0; n < batchsize; n++) {
    for (int c = 0; c < channels; ++c) {
      scalar_t out_value = odata[n][c][output_y][output_x];
      for (int y = 0; y < ysize; y++) {
        for (int x = 0; x < xsize; x++) {
          upsample_increment_value_bounded<scalar_t, accscalar_t>(
              idata,
              n,
              c,
              input_height,
              input_width,
              ymin + y,
              xmin + x,
              wx[x] * wy[y] * out_value);
        }
      }
    }
  }
}

template <int interp_size>
static void upsample_gen2d_backward_out_cuda_template(
    const Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  // Copied and adapted from
  // UpSampleBicubic2d.cu::upsample_bicubic2d_backward_out_cuda_template
  TensorArg grad_input_arg{grad_input, "grad_input", 1},
      grad_output_arg{grad_output_, "grad_output_", 2};
  checkAllSameGPU(
      "upsample_gen2d_backward_out_cuda", {grad_output_arg, grad_input_arg});

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input_size[0];
  int channels = input_size[1];
  int input_height = input_size[2];
  int input_width = input_size[3];

  Tensor grad_output = grad_output_.contiguous();

  grad_input.zero_();

  const int num_kernels = output_height * output_width;
  const int num_threads = std::min(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(), "upsample_gen2d_backward_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = grad_input.packed_accessor64<scalar_t, 4>();
        auto odata = grad_output.packed_accessor64<scalar_t, 4>();

        const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
            input_height, output_height, align_corners, scales_h);
        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners, scales_w);

        // We are using static buffer memory of 256 * sizeof(float) per thread
        // to store weights. Size of weights array is
        // interp_size = scale * 2 + 1 for bilinear mode
        TORCH_CHECK(
            rheight < (255 / interp_size),
            "Max supported scale factor is 127 (bilinear), 63 (bicubic)");
        TORCH_CHECK(
            rwidth < (255 / interp_size),
            "Max supported scale factor is 127 (bilinear), 63 (bicubic)");

        upsample_gen2d_backward_out_frame<scalar_t, accscalar_t, interp_size>
            <<<cuda::ATenCeilDiv(num_kernels, num_threads),
               num_threads,
               0,
               stream>>>(
                num_kernels, rheight, rwidth, align_corners, idata, odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

} // namespace internal_upsample
} // namespace native
} // namespace at

namespace vision {
namespace ops {

namespace {

// Copied from "UpSample.h" as we can not use UpSample.h with UpSample.cuh
static std::array<int64_t, 4> upsample_2d_common_check(
    at::IntArrayRef input_size,
    at::IntArrayRef output_size) {
  TORCH_CHECK(
      output_size.size() == 2,
      "It is expected output_size equals to 2, but got size ",
      output_size.size());

  TORCH_CHECK(
      input_size.size() == 4,
      "It is expected input_size equals to 4, but got size ",
      input_size.size());

  int64_t output_height = output_size[0];
  int64_t output_width = output_size[1];

  int64_t nbatch = input_size[0];
  int64_t channels = input_size[1];
  int64_t input_height = input_size[2];
  int64_t input_width = input_size[3];

  TORCH_CHECK(
      input_height > 0 && input_width > 0 && output_height > 0 &&
          output_width > 0,
      "Input and output sizes should be greater than 0,"
      " but got input (H: ",
      input_height,
      ", W: ",
      input_width,
      ") output (H: ",
      output_height,
      ", W: ",
      output_width,
      ")");

  return {nbatch, channels, output_height, output_width};
}

template <int interp_size>
at::Tensor interpolate_gen2d_aa_forward_kernel(
    const at::Tensor& input,
    at::IntArrayRef output_size,
    bool align_corners) {
  c10::optional<c10::ArrayRef<double>> scale_factors = {};

  // Copied from UpSampleBilinear2d.cpp
  auto output = at::empty({0}, input.options());
  auto osize = at::native::upsample::compute_output_size(
      input.sizes(), output_size, scale_factors);
  auto scale_h = at::native::upsample_cuda::get_scale_value(scale_factors, 0);
  auto scale_w = at::native::upsample_cuda::get_scale_value(scale_factors, 1);

  auto full_output_size = upsample_2d_common_check(input.sizes(), osize);

  // Allow for empty batch size but not other dimensions
  TORCH_CHECK(
      input.numel() != 0 ||
          c10::multiply_integers(
              input.sizes().begin() + 1, input.sizes().end()),
      "Non-empty 4D data tensor expected but got a tensor with sizes ",
      input.sizes());

  output.resize_(full_output_size, input.suggest_memory_format());

  at::native::internal_upsample::upsample_gen2d_out_cuda_template<interp_size>(
      output,
      input,
      {full_output_size[2], full_output_size[3]},
      align_corners,
      scale_h,
      scale_w);
  return output;
}

template <int interp_size>
at::Tensor interpolate_gen2d_aa_backward_kernel(
    const at::Tensor& grad_output,
    at::IntArrayRef output_size,
    at::IntArrayRef input_size,
    bool align_corners) {
  c10::optional<c10::ArrayRef<double>> scale_factors = {};

  // Copied from UpSampleBicubic2d.cpp::upsample_bicubic2d_backward
  auto grad_input = at::empty({0}, grad_output.options());
  auto osize = at::native::upsample::compute_output_size(
      input_size, output_size, scale_factors);
  auto scale_h = at::native::upsample_cuda::get_scale_value(scale_factors, 0);
  auto scale_w = at::native::upsample_cuda::get_scale_value(scale_factors, 1);

  auto full_output_size = upsample_2d_common_check(input_size, osize);

  TORCH_CHECK(
      grad_output.dim() == 4,
      "Expected grad_output to be a tensor of dimension 4 but got: dimension ",
      grad_output.dim());

  for (int i = 0; i < 4; ++i) {
    TORCH_CHECK(
        grad_output.size(i) == full_output_size[i],
        "Expected grad_output to have the same shape as output;",
        " output.size(",
        i,
        ") = ",
        full_output_size[i],
        " but got grad_output.size(",
        i,
        ") = ",
        grad_output.size(i));
  }

  grad_input.resize_(input_size, grad_output.suggest_memory_format());

  at::native::internal_upsample::upsample_gen2d_backward_out_cuda_template<
      interp_size>(
      grad_input,
      grad_output,
      {full_output_size[2], full_output_size[3]},
      input_size,
      align_corners,
      scale_h,
      scale_w);
  return grad_input;
}

at::Tensor interpolate_bilinear2d_aa_forward_kernel(
    const at::Tensor& input,
    at::IntArrayRef output_size,
    bool align_corners) {
  return interpolate_gen2d_aa_forward_kernel<2>(
      input, output_size, align_corners);
}

at::Tensor interpolate_bicubic2d_aa_forward_kernel(
    const at::Tensor& input,
    at::IntArrayRef output_size,
    bool align_corners) {
  return interpolate_gen2d_aa_forward_kernel<4>(
      input, output_size, align_corners);
}

at::Tensor interpolate_bilinear2d_aa_backward_kernel(
    const at::Tensor& grad_output,
    at::IntArrayRef output_size,
    at::IntArrayRef input_size,
    bool align_corners) {
  return interpolate_gen2d_aa_backward_kernel<2>(
      grad_output, output_size, input_size, align_corners);
}

at::Tensor interpolate_bicubic2d_aa_backward_kernel(
    const at::Tensor& grad_output,
    at::IntArrayRef output_size,
    at::IntArrayRef input_size,
    bool align_corners) {
  return interpolate_gen2d_aa_backward_kernel<4>(
      grad_output, output_size, input_size, align_corners);
}

} // namespace

TORCH_LIBRARY_IMPL(torchvision, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("torchvision::_interpolate_bilinear2d_aa"),
      TORCH_FN(interpolate_bilinear2d_aa_forward_kernel));
  m.impl(
      TORCH_SELECTIVE_NAME("torchvision::_interpolate_bicubic2d_aa"),
      TORCH_FN(interpolate_bicubic2d_aa_forward_kernel));
  m.impl(
      TORCH_SELECTIVE_NAME("torchvision::_interpolate_bilinear2d_aa_backward"),
      TORCH_FN(interpolate_bilinear2d_aa_backward_kernel));
  m.impl(
      TORCH_SELECTIVE_NAME("torchvision::_interpolate_bicubic2d_aa_backward"),
      TORCH_FN(interpolate_bicubic2d_aa_backward_kernel));
}

} // namespace ops
} // namespace vision
